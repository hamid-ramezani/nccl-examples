#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <stdlib.h>
#include <hip/hip_runtime_api.h>


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{
  ncclComm_t comms[4];

  //managing 4 devices
  int nDev = 4;
  //int size = 32*1024*1024;
 
  int size = 1;
  int devs[4] = { 0, 1, 2, 3 };


  //allocating and initializing device buffers
  int** sendbuff = (int**)malloc(nDev * sizeof(int8_t*));
  int** recvbuff = (int**)malloc(nDev * sizeof(int8_t*));
  hipStream_t* s = (hipStream_t*)malloc(nDev * sizeof(hipStream_t));
  
  int8_t* h_sendbuff = (int8_t*)malloc(size * sizeof(int8_t));
  int8_t* h_recvbuff = (int8_t*)malloc(size * sizeof(int8_t));
  
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc((void**)sendbuff + i, size * sizeof(int8_t)));
    CUDACHECK(hipMalloc((void**)recvbuff + i, size * sizeof(int8_t)));
    CUDACHECK(hipMemset(sendbuff[i], 13, size * sizeof(int8_t)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(int8_t)));
    CUDACHECK(hipStreamCreate(s+i));
    hipDeviceSynchronize();
  }

   hipMemcpy(h_sendbuff,sendbuff[0],size * sizeof(int8_t),hipMemcpyDeviceToHost);
   hipDeviceSynchronize();
   printf("%i",h_sendbuff[0]);
   printf("\n");


  //initializing NCCL
  //calling NCCL communication API. Group API is required when using
  //multiple devices per thread
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i){
   // NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum, comms[i], s[i]));
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclInt8, ncclSum, comms[i], s[i]));
  }

  NCCLCHECK(ncclGroupEnd());



   hipMemcpy(h_recvbuff,recvbuff[0],size * sizeof(int8_t),hipMemcpyDeviceToHost);
   hipDeviceSynchronize();
   printf("%i",h_recvbuff[0]);
   printf("\n");
 

 //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }



  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }


  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);


  printf("Success \n");
  return 0;
}

