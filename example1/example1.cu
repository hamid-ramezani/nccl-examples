#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{

  ncclComm_t comms[4];

  //managing 4 devices
  int nDev = 4;

  //int size = 256*1024*1024;
  //int size = 32*32*32;
  int size = 8;

  int devs[4] = { 0, 1, 2, 3 };
  //int devs[8] = { 0, 1, 2, 3, 4, 5, 6, 7 };
  //size_t  heapSize = 1024 * 1024 * 1024;
  //hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize);


  //allocating and initializing device buffers
  //int8_t** sendbuff = (int8_t**)malloc(nDev * sizeof(int8_t*));
  //int8_t** recvbuff = (int8_t**)malloc(nDev * sizeof(int8_t*));
  //int8_t** tempbuff = (int8_t**)malloc(nDev * sizeof(int8_t*));
  
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  //float** tempbuff = (float**)malloc(nDev * sizeof(float*));
  
  hipStream_t* s = (hipStream_t*)malloc(nDev * sizeof(hipStream_t));
  
  //int8_t* h_sendbuff = (int8_t*)malloc(size * sizeof(int8_t));
  //int8_t* h_recvbuff = (int8_t*)malloc(size * sizeof(int8_t));
  
  float* h_sendbuff = (float*)malloc(size * sizeof(float));
  float* h_recvbuff = (float*)malloc(size * sizeof(float));
  
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    //CUDACHECK(hipMalloc((void**)sendbuff + i, size * sizeof(int8_t)));
    //CUDACHECK(hipMalloc((void**)recvbuff + i, size * sizeof(int8_t)));
    //CUDACHECK(hipMemset(sendbuff[i], 13, size * sizeof(int8_t)));
    //CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(int8_t)));

    //CUDACHECK(hipMalloc((void**)tempbuff + i, size * sizeof(int8_t)));
    //CUDACHECK(hipMemset(tempbuff[i], 0, size * sizeof(int8_t)));
  
  
    CUDACHECK(hipMalloc((void**)sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc((void**)recvbuff + i, size * sizeof(float)));

    float fill_value1 = 2.4;
    thrust::device_ptr<float> dev_ptr1(sendbuff[i]);
    thrust::fill(dev_ptr1, dev_ptr1 + size, fill_value1);

    float fill_value2 = 0;
    thrust::device_ptr<float> dev_ptr2(recvbuff[i]);
    thrust::fill(dev_ptr2, dev_ptr2 + size, fill_value2);



    //CUDACHECK(hipMemset(sendbuff[i], 2., size * sizeof(float)));
    //CUDACHECK(hipMemset(recvbuff[i], 0., size * sizeof(float)));

    //CUDACHECK(hipMalloc((void**)tempbuff + i, size * sizeof(float)));
    //CUDACHECK(hipMemset(tempbuff[i], 0, size * sizeof(float)));
  
    CUDACHECK(hipStreamCreate(s+i));
    CUDACHECK(hipDeviceSynchronize());
  }

   //CUDACHECK(hipMemcpy(h_sendbuff,sendbuff[0],size * sizeof(int8_t),hipMemcpyDeviceToHost));
   CUDACHECK(hipMemcpy(h_sendbuff,sendbuff[0],size * sizeof(float),hipMemcpyDeviceToHost));
   CUDACHECK(hipDeviceSynchronize());
   


  //initializing NCCL
  //calling NCCL communication API. Group API is required when using
  //multiple devices per thread
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i){
    //NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclInt8 , ncclSum, comms[i], s[i]));
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum, comms[i], s[i]));
    //NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], (void*)tempbuff[i], size, ncclInt8 , ncclSum, comms[i], s[i]));
    //NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], (void*)tempbuff[i], size, ncclFloat , ncclSum, comms[i], s[i]));
  }

  NCCLCHECK(ncclGroupEnd());


  //CUDACHECK(hipMemcpy(h_recvbuff,recvbuff[0],size * sizeof(int8_t),hipMemcpyDeviceToHost));
  CUDACHECK(hipMemcpy(h_recvbuff,recvbuff[0],size * sizeof(float),hipMemcpyDeviceToHost));
  CUDACHECK(hipDeviceSynchronize());

   //for (int i = 0; i< size; ++i) {
   //  printf("%i\n",h_sendbuff[i]);
   //}
   
   for (int i = 0; i< size; ++i) {
     printf("%f\n",h_sendbuff[i]);
   }

   //for (int i = 0; i< size; ++i) {
   //  printf("%i\n",h_recvbuff[i]);
   //}
 
   for (int i = 0; i< size; ++i) {
     printf("%f\n",h_recvbuff[i]);
   }

   //printf("the first element of the array is: %d \n", h_recvbuff[0]);
   //int count = 0;
   //for(int i=0; i<size; ++i){
   //  if(h_recvbuff[i] != 52){
   //     count++; 
   //     //printf("h_recvbuff[%d] = %d \n", i, h_recvbuff[i]);
   //  }
   //}
   //printf("count = %d \n", count);


 //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }



  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
    //CUDACHECK(hipFree(tempbuff[i]));
  }


  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);


  printf("Success \n");
  return 0;
}
